
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <stdio.h>
#include <stdlib.h>
#include <chrono>

#define SAFE_CALL(CallInstruction) { hipError_t hipError_t = CallInstruction; if (hipError_t != hipSuccess) { printf("CUDA error: %s at call %s", hipGetErrorString(hipError_t), #CallInstruction); exit(0); } }

__global__ void addKernel(double* c, double* a, double* b, unsigned int size) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < size; i += blockDim.x) {
        c[i] = a[i] + b[i];
    }
}

double calculateForParameters(int gridSize, int blockSize, int n, int maxTries) {
    double time = 0.0;
    for (int k = 0; k < maxTries; ++k) {
        int n2b = n * sizeof(double);

        double* a = (double*) calloc(n, sizeof(double));
        double* b = (double*) calloc(n, sizeof(double));
        double* c = (double*) calloc(n, sizeof(double));

        for (int i = 0; i < n; ++i) {
            a[i] = i;
            b[i] = i;
        }
        double* aDevice = NULL;
        double* bDevice = NULL;
        double* cDevice = NULL;
        SAFE_CALL(hipMalloc((void**) &aDevice, n2b));
        SAFE_CALL(hipMalloc((void**) &bDevice, n2b));
        SAFE_CALL(hipMalloc((void**) &cDevice, n2b));
        hipEvent_t start, stop;
        SAFE_CALL(hipEventCreate(&start));
        SAFE_CALL(hipEventCreate(&stop));
        SAFE_CALL(hipMemcpy(aDevice, a, n2b, hipMemcpyHostToDevice));
        SAFE_CALL(hipMemcpy(bDevice, b, n2b, hipMemcpyHostToDevice));
        SAFE_CALL(hipEventRecord(start, 0));
        addKernel <<< gridSize, blockSize >>> (cDevice, aDevice, bDevice, n);
        SAFE_CALL(hipDeviceSynchronize());
        SAFE_CALL(hipGetLastError());
        SAFE_CALL(hipMemcpy(c, cDevice, n2b, hipMemcpyDeviceToHost));
        SAFE_CALL(hipEventRecord(stop, 0));
        float gpuTime = 0.0f;
        SAFE_CALL(hipDeviceSynchronize());
        SAFE_CALL(hipEventElapsedTime(&gpuTime, start, stop));

        time += gpuTime * 1000;
        hipEventDestroy(start);
        hipEventDestroy(stop);
        hipFree(aDevice);
        hipFree(bDevice);
        hipFree(cDevice);
        free(a);
        free(b);
        free(c);
    }
    return time / maxTries;
}

double calculateConsistent(int n) {
    double* a = (double*) calloc(n, sizeof(double));
    double* b = (double*) calloc(n, sizeof(double));
    double* c = (double*) calloc(n, sizeof(double));

    for (int i = 0; i < n; ++i) {
        a[i] = i;
        b[i] = i;
    }

    auto start = std::chrono::system_clock::now();

    for (int i = 0; i < n; ++i) {
        c[i] = a[i] + b[i];
    }

    auto end = std::chrono::system_clock::now();
    double time = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();

    free(a);
    free(b);
    free(c);
    return time;
}

int main(int argc, char* argv[]) {
    int maxTries = 12;
    int startN = 6000000;
    int nStep = 5;
    int gridsAndBlocks[6][2] = {
        { 2048, 1024 },
        { 1024, 1024 },
        { 2048, 512 },
        { 1024, 512 },
        { 2048, 256 },
        { 1024, 256 }
    };

    for (int n = startN; n > 240000 - 1 ; n /= nStep) {
        printf("n = %d\n", n);
        printf("Consistent: ");
        double cTime = 0.0;
        for (int k = 0; k < maxTries; ++k) {
            cTime += calculateConsistent(n);
        }
        printf("%.4f\n", cTime / maxTries);
        printf("CUDA: ");
        for (int j = 0; j < 6; ++j) {
            printf("[GridDim, BlockDim] = [%d, %d]: ", gridsAndBlocks[j][0], gridsAndBlocks[j][1]);
            double time = calculateForParameters(gridsAndBlocks[j][0], gridsAndBlocks[j][1], n, maxTries);
            printf("%.4f\n", time);
        }
    }
    return 0;
}
