#include "hip/hip_runtime.h"
#include <malloc.h>
#include <stdio.h>
#include <stdlib.h>
#include <chrono>

#define SAFE_CALL(CallInstruction) { hipError_t hipError_t = CallInstruction; if (hipError_t != hipSuccess) { printf("CUDA error: %s at call %s", hipGetErrorString(hipError_t), #CallInstruction); exit(0); } }

__global__ void addKernel(int* c, int* a, int* b, unsigned int size) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < size; i += blockDim.x) {
        c[i] = a[i] + b[i];
    }
}

void calculateForParameters(int gridSize, int blockSize, int n, int maxTries) {
    double gpuAverage = 0.0;
    double cpuAverage = 0.0;
    for (int k = 0; k < maxTries; ++k) {
        int n2b = n * sizeof(int);

        int* a = (int*) calloc(n, sizeof(int));
        int* b = (int*) calloc(n, sizeof(int));
        int* c = (int*) calloc(n, sizeof(int));

        for (int i = 0; i < n; ++i) {
            a[i] = i;
            b[i] = i;
        }
        int* aDevice = NULL;
        int* bDevice = NULL;
        int* cDevice = NULL;
        SAFE_CALL(hipMalloc((void**) &aDevice, n2b));
        SAFE_CALL(hipMalloc((void**) &bDevice, n2b));
        SAFE_CALL(hipMalloc((void**) &cDevice, n2b));
        hipEvent_t start, stop;
        SAFE_CALL(hipEventCreate(&start));
        SAFE_CALL(hipEventCreate(&stop));
        SAFE_CALL(hipMemcpy(aDevice, a, n2b, hipMemcpyHostToDevice));
        SAFE_CALL(hipMemcpy(bDevice, b, n2b, hipMemcpyHostToDevice));
        double startTime = clock();
        for (int i = 0; i < n; ++i) {
            c[i] = a[i] + b[i];
        }
        double cpuTime = (clock() - startTime) / CLOCKS_PER_SEC * 1000 * 1000;
        SAFE_CALL(hipEventRecord(start, 0));
        addKernel <<< gridSize, blockSize >>> (cDevice, aDevice, bDevice, n);
        SAFE_CALL(hipDeviceSynchronize());
        SAFE_CALL(hipGetLastError());
        SAFE_CALL(hipMemcpy(c, cDevice, n2b, hipMemcpyDeviceToHost));
        SAFE_CALL(hipEventRecord(stop, 0));
        float gpuTime = 0.0f;
        SAFE_CALL(hipEventElapsedTime(&gpuTime, start, stop));

        double time = gpuTime * 1000;
        hipEventDestroy(start);
        hipEventDestroy(stop);
        hipFree(aDevice);
        hipFree(bDevice);
        hipFree(cDevice);
        free(a);
        free(b);
        free(c);
        printf("cpu: %f, gpu: %f\n", cpuTime, gpuTime);
        return time;
    }
}

double calculateConsistent(int n) {
    int* a = (int*) calloc(n, sizeof(int));
    int* b = (int*) calloc(n, sizeof(int));
    int* c = (int*) calloc(n, sizeof(int));

    for (int i = 0; i < n; ++i) {
        a[i] = i;
        b[i] = i;
    }

    auto start = std::chrono::system_clock::now();

    for (int i = 0; i < n; ++i) {
        c[i] = a[i] + b[i];
    }

    auto end = std::chrono::system_clock::now();
    double time = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();

    free(a);
    free(b);
    free(c);
    return time;
}

int main(int argc, char* argv[]) {
    int maxTries = 12;
    int startN = 6000000;
    int nStep = 5;
    int gridsAndBlocks[6][2] = {
        { 2048, 1024 },
        { 1024, 1024 },
        { 2048, 512 },
        { 1024, 512 },
        { 2048, 256 },
        { 1024, 256 }
    };

    for (int n = startN; n > 240000 - 1 ; n /= nStep) {
        printf("n = %d\n", n);
        printf("Consistent: ");
        double cTime = 0.0;
        for (int k = 0; k < maxTries; ++k) {
            cTime += calculateConsistent(n);
        }
        printf("%.4f\n", cTime / 12);
        for (int j = 0; j < 6; ++j) {
            printf("[GridDim, BlockDim] = [%d, %d]: ", gridsAndBlocks[j][0], gridsAndBlocks[j][1]);
            double time = 0.0;
            time += calculateForParameters(gridsAndBlocks[j][0], gridsAndBlocks[j][1], n);
            printf("%.4f\n", time / 12);
        }
    }
    return 0;
}
