#include "hip/hip_runtime.h"
%%cuda --name lab5curand.cu

#include <cstdlib>
#include <hiprand.h>
#include <hipblas.h>
#include <iostream>
#include <ctime>

#define SAFE_CALL(CallInstruction) { hipError_t hipError_t = CallInstruction; if (hipError_t != hipSuccess) { printf("CUDA error: %s at call %s", hipGetErrorString(hipError_t), #CallInstruction); exit(0); } }
#define IDX2C(i,j,ld) (((j)*(ld))+(i))

using namespace std;

void print_matrix(float* matrix, int rows, int cols) {
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            printf("%f ", matrix[i + j * rows]);
        }
        printf("\n");
    }
    printf("\n");
}

void matrix_mult(float* A, float* B, float* C, int N)
{
    int i, j, k;
    for (i = 0; i < N; i++) {
        for (j = 0; j < N; j++) {
            C[IDX2C(i, j, N)] = 0;
            for (k = 0; k < N; k++) {
                C[IDX2C(i, j, N)] += A[i + k * N] * B[k + j * N];
            }
        }
    }
}

void fill_matrix(float* matrix, int rows, int cols) {
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            matrix[i + j * rows] = (rand() + 0.0) / RAND_MAX;
        }
    }
}

void GPU_fill_rand(float *A, int nr_rows_A, int nr_cols_A) {
    hiprandGenerator_t prng;
    hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock());
    hiprandGenerateUniform(prng, A, nr_rows_A * nr_cols_A);
}

double calculateGPU(const float *A, const float *B, float *C, const int m, const int k, const int n) {
    int lda=m,ldb=k,ldc=m;
    const float alf = 1;
    const float bet = 0;
    const float *alpha = &alf;
    const float *beta = &bet;
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipEvent_t start, stop;
    SAFE_CALL(hipEventCreate(&start));
    SAFE_CALL(hipEventCreate(&stop));

    SAFE_CALL(hipEventRecord(start, 0));
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
    SAFE_CALL(hipEventRecord(stop, 0));

    hipblasDestroy(handle);

    float gpuTime = 0.0f;
    SAFE_CALL(hipEventElapsedTime(&gpuTime, start, stop));

    hipEventDestroy(start);
    hipEventDestroy(stop);

    double time = gpuTime * 1000;
    return time;
}

void testGPU(int squareSize) {
// Заполнение матриц на GPU
    int nr_rows_A, nr_cols_A, nr_rows_B, nr_cols_B, nr_rows_C, nr_cols_C;
    nr_rows_A = nr_cols_A = nr_rows_B = nr_cols_B = nr_rows_C = nr_cols_C = squareSize;

    float *d_A, *d_B, *d_C, *C;

    SAFE_CALL(hipMalloc(&d_A, nr_rows_A * nr_cols_A * sizeof(float)));
    SAFE_CALL(hipMalloc(&d_B, nr_rows_B * nr_cols_B * sizeof(float)));
    SAFE_CALL(hipMalloc(&d_C, nr_rows_C * nr_cols_C * sizeof(float)));
    C = (float *)malloc(nr_rows_A * nr_cols_A * sizeof(float));

    hipEvent_t start, stop;
    SAFE_CALL(hipEventCreate(&start));
    SAFE_CALL(hipEventCreate(&stop));
    SAFE_CALL(hipEventRecord(start, 0));
    GPU_fill_rand(d_A, nr_rows_A, nr_cols_A);
    GPU_fill_rand(d_B, nr_rows_B, nr_cols_B);
    SAFE_CALL(hipMemcpy(c, d_C, nr_rows_C * nr_cols_C * sizeof(float), hipMemcpyDeviceToHost));
    SAFE_CALL(hipDeviceSynchronize());
    SAFE_CALL(hipEventRecord(stop, 0));
    float gpuTime = 0.0f;
    SAFE_CALL(hipEventElapsedTime(&gpuTime, start, stop));
    cout << "GPU create: " << gpuTime * 1000 << " milliseconds\n";
    hipEventDestroy(start);
    hipEventDestroy(stop);

    double gpuTime = calculateGPU(d_A, d_B, d_C, nr_rows_A, nr_cols_A, nr_cols_B);
    cout << "calculateGPU time = " << gpuTime << " microseconds";
    print_matrix(C, nr_rows_C, nr_cols_C);

    free(C);
    SAFE_CALL(hipFree(d_A));
    SAFE_CALL(hipFree(d_B));
    SAFE_CALL(hipFree(d_C));
}

void testCPU(int squareSize) {
    int nr_rows_A, nr_cols_A, nr_rows_B, nr_cols_B, nr_rows_C, nr_cols_C;
    nr_rows_A = nr_cols_A = nr_rows_B = nr_cols_B = nr_rows_C = nr_cols_C = squareSize;
    float *d_A, *d_B;
    float *h_A = (float *)malloc(nr_rows_A * nr_cols_A * sizeof(float));
    float *h_B = (float *)malloc(nr_rows_B * nr_cols_B * sizeof(float));
    float *h_C = (float *)malloc(nr_rows_C * nr_cols_C * sizeof(float));

    SAFE_CALL(hipMalloc(&d_A, nr_rows_A * nr_cols_A * sizeof(float)));
    SAFE_CALL(hipMalloc(&d_B, nr_rows_B * nr_cols_B * sizeof(float)));

    double start_consistent = clock();
    fill_matrix(h_A, nr_rows_A, nr_cols_A);
    fill_matrix(h_B, nr_rows_B, nr_cols_B);
    SAFE_CALL(hipMemcpy(h_A, d_B, nr_rows_A * nr_cols_A * sizeof(float), hipMemcpyDeviceToHost));
    SAFE_CALL(hipMemcpy(h_B, d_B, nr_rows_B * nr_cols_B * sizeof(float), hipMemcpyDeviceToHost));
    SAFE_CALL(hipDeviceSynchronize());
    double cpuGenerateTime = ((clock() - start_consistent) / CLOCKS_PER_SEC) * 1000 * 1000;
    cout << "GPU create: " << cpuGenerateTime <<  " milliseconds\n";

    start_consistent = clock();
    matrix_mult(h_A, h_B, h_C, nr_rows_A);
    double cpuTime = ((clock() - start_consistent) / CLOCKS_PER_SEC) * 1000 * 1000;
    cout << "calculateCPU time: " << cpuGenerateTime << " milliseconds\n";
    print_matrix(h_C, nr_rows_C, nr_cols_C);

    free(h_A);
    free(h_B);
    free(h_C);

    SAFE_CALL(hipFree(d_A));
    SAFE_CALL(hipFree(d_B));
}

int main() {
    for (int size = 100; size <= 2500; size *= 500) {
        testCPU(size);
        testGPU(size);
    }
    return 0;
}
